#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <stdlib.h>
#include <random>

#include "../gif-h/gif.h"

#define PIXEL_DEPTH 4
#define BLOCK_SIZE 256
#define DELAY 10
#define SIMULATION_STEPS 100
#define SIMULATION_SEED 0xFFAABBEE
#define SIMULATION_SIZE 256

__device__
unsigned idx(int u, int v, unsigned size) {
  u = (u + size) % size;
  v = (v + size) % size;
  return v * size + u;
}

__device__
void set(uint8_t *w, int i, bool alive) {
  uint8_t color = alive ? 255 : 0;
  w[i * PIXEL_DEPTH + 0] = color;
  w[i * PIXEL_DEPTH + 1] = color;
  w[i * PIXEL_DEPTH + 2] = color;
  w[i * PIXEL_DEPTH + 3] = color;
}

__global__
void gameOfLifeStep(unsigned n, uint8_t *r, uint8_t *w, unsigned size) {
  unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    int v = i / size;
    int u = i - v * size;

    unsigned neighbors = 0;
    for (int dv = -1; dv <= 1; dv++) {
      for (int du = -1; du <= 1; du++) {
        if (r[idx(u + du, v + dv, size) * PIXEL_DEPTH] == UINT8_MAX) {
          neighbors++;
        }
      }
    }

    bool alive = r[i * PIXEL_DEPTH] == UINT8_MAX;

    set(w, i, alive ? neighbors == 2 || neighbors == 3 : neighbors == 3);
  }
}

int main(int argc, char *argv[]) {

  std::mt19937 generator (SIMULATION_SEED);

  unsigned pixCount = SIMULATION_SIZE * SIMULATION_SIZE;

  // use two buffers to avoid contention
  uint8_t *buffer[2];

  // allocate memory
  hipMallocManaged(&buffer[0], PIXEL_DEPTH * pixCount * sizeof(uint8_t));
  hipMallocManaged(&buffer[1], PIXEL_DEPTH * pixCount * sizeof(uint8_t));

  // init the buffers
  for (int j = 0; j < pixCount; j++) {
    unsigned color = generator() % 2 == 1
      ? UINT8_MAX
      : 0;
    for (int k = 0; k < PIXEL_DEPTH; k++) {
      buffer[0][j * PIXEL_DEPTH + k] = color;
    }
  }

  GifWriter gif;

  GifBegin(&gif, "out.gif", SIMULATION_SIZE, SIMULATION_SIZE, DELAY);

  unsigned current = 0;
  for (unsigned i = 0; i < SIMULATION_STEPS; i++) {
    // current and next are the two buffers; we always write to the other buffer to avoid read-write contention

    unsigned next = (current + 1) % 2;

    // write out the frame
    GifWriteFrame(&gif, buffer[current], SIMULATION_SIZE, SIMULATION_SIZE, DELAY);

    // simulate
    gameOfLifeStep<<<(pixCount + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE>>>(pixCount, buffer[current], buffer[next], SIMULATION_SIZE);

    // synchronize
    hipDeviceSynchronize();

    // swap buffers
    current = next;
  }

  GifWriteFrame(&gif, buffer[current], SIMULATION_SIZE, SIMULATION_SIZE, DELAY);

  GifEnd(&gif);

  return 0;
}